#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"
#include "strided_copy.h"

__thread int mem_stride_copy_gridsize = 1;
__thread int mem_stride_copy_blocksize = 1;
__thread int local_size = 1;
__thread void* scratch_buff = NULL;

void print_header() {
  PRINT("# %10s  %12s  %8s  %6s            out-of-place                       in-place          \n", "", "", "", "");
  PRINT("# %10s  %12s  %8s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %8s  %6s", size, count, typeName, opName);
}

void AlltoAll2DHGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = (count/nranks)*nranks;
  *recvcount = (count/nranks)*nranks;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = count/nranks;
}

testResult_t AlltoAll2DHInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;
  // PRINT("\ninit data, nranks %d, sendcount %ld, recvcount %ld. sendBytes %ld, expectedBytes %ld \n", nranks, sendcount, recvcount, args->sendBytes, args->expectedBytes);

  CUDACHECK(hipOccupancyMaxPotentialBlockSize(&mem_stride_copy_gridsize, &mem_stride_copy_blocksize, memStrideCopyUInt4Kernel));
  CUDACHECK(hipGetDeviceCount(&local_size));

  for (int i=0; i<args->nGpus; i++) {
    char* str = getenv("NCCL_TESTS_DEVICE\n");
    int gpuid = str ? atoi(str) : args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, type, rep, rank));
    for (int j=0; j<nranks; j++) {
      TESTCHECK(InitData(((char*)args->expected[i])+args->sendBytes/nranks*j, sendcount/nranks, type, rep+rank*sendcount/nranks, j));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place alltoall
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void AlltoAll2DHGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  // PRINT("nranks %d, count %ld, typesize %d", nranks, count, typesize);
  double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks-1))/((double)(nranks));
  *busBw = baseBw * factor;
}

testResult_t AlltoAll2DHRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  CUDACHECK(hipGetDeviceCount(&local_size));
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  int local_rank;
  NCCLCHECK(ncclCommCuDevice(comm, &local_rank));
  size_t rankOffset = count * wordSize(type);
  if (count == 0) return testSuccess;
  PRINT("RUN COLL send count %ld local_size %d\n", count, local_size);

#if NCCL_MAJOR < 2 || NCCL_MINOR < 7
  printf("NCCL 2.7 or later is needed for alltoall. This test was compiled with %d.%d.\n", NCCL_MAJOR, NCCL_MINOR);
  return testNcclError;
#elif SCCL_SUPPORT
  NCCLCHECK(msccl2DAllToAll(sendbuff, recvbuff, count, type, comm, stream));
  return testSuccess;
#else
  if (nRanks % local_size != 0) {
    printf("AlltoAll: nranks %d is not a multiple of local_size %d\n", nRanks, local_size);
    return testNcclError;
  }
  int nnodes = nRanks / local_size;
  if (!(local_size == 1 || nnodes == 1)) {
    int node_rank = rank / local_size;
    PRINT("node rank %d\n", node_rank);

    size_t slice_size = count * wordSize(type) / nRanks;
    size_t slice_size_uint4 = slice_size / sizeof(uint4);

    // phase 0. per-gpu (ngpus) stride copy
    if (slice_size < sizeof(uint4)) {
      memStrideCopyCharKernel<<<mem_stride_copy_gridsize, mem_stride_copy_blocksize, 0, stream>>>(
        (char*)scratch_buff, (char*)sendbuff, slice_size, local_size, nnodes);
    } else {
      memStrideCopyUInt4Kernel<<<mem_stride_copy_gridsize, mem_stride_copy_blocksize, 0, stream>>>(
        (uint4*)scratch_buff, (uint4*)sendbuff, slice_size_uint4, local_size, nnodes);
    }

    // phase 1. intra-node alltoall
    NCCLCHECK(ncclGroupStart());
    for (int g = 0; g < local_size; g++) {
      NCCLCHECK(ncclSend(((char*)scratch_buff) + g * nnodes * slice_size, nnodes * slice_size, ncclInt8, g + node_rank * local_size, comm, stream));
      NCCLCHECK(ncclRecv(((char*)sendbuff) + g * nnodes * slice_size, nnodes * slice_size, ncclInt8, g + node_rank * local_size, comm, stream));
    }
    NCCLCHECK(ncclGroupEnd());

    // phase 2. per-gpu (nnodes) stride copy
    if (slice_size < sizeof(uint4)) {
      memStrideCopyCharKernel<<<mem_stride_copy_gridsize, mem_stride_copy_blocksize, 0, stream>>>(
        (char*)scratch_buff, (char*)sendbuff, slice_size, nnodes, local_size);
    } else {
      memStrideCopyUInt4Kernel<<<mem_stride_copy_gridsize, mem_stride_copy_blocksize, 0, stream>>>(
        (uint4*)scratch_buff, (uint4*)sendbuff, slice_size_uint4, nnodes, local_size);
    }

    // phase 3. inter-node alltoall
     NCCLCHECK(ncclGroupStart());
    for (int n = 0; n < nnodes; n++) {
      NCCLCHECK(ncclSend(((char*)scratch_buff) + n * local_size * slice_size, local_size * slice_size, ncclInt8, n * local_size + local_rank, comm, stream));
      NCCLCHECK(ncclRecv(((char*)sendbuff) + n * local_size * slice_size, local_size * slice_size, ncclInt8, n * local_size + local_rank, comm, stream));
    }
    NCCLCHECK(ncclGroupEnd());
    local_size = 1;
    return testSuccess;
  } else {
    NCCLCHECK(ncclGroupStart());
    for (int r=0; r<nRanks; r++) {
      NCCLCHECK(ncclSend(((char*)sendbuff)+r*rankOffset, count, type, r, comm, stream));
      NCCLCHECK(ncclRecv(((char*)recvbuff)+r*rankOffset, count, type, r, comm, stream));
    }
    NCCLCHECK(ncclGroupEnd());
    local_size = 1;
    return testSuccess;
  }
#endif
}

struct testColl alltoAll2DHTest = {
  "AlltoAll2DH",
  AlltoAll2DHGetCollByteCount,
  AlltoAll2DHInitData,
  AlltoAll2DHGetBw,
  AlltoAll2DHRunColl
};

void AlltoAll2DHGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AlltoAll2DHGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t AlltoAll2DHRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &alltoAll2DHTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  CUDACHECK(hipMalloc(&scratch_buff, args->maxbytes));

  for (int i=0; i<type_count; i++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "", -1));
  }

  CUDACHECK(hipFree(scratch_buff));
  return testSuccess;
}

struct testEngine alltoAll2DHEngine = {
  AlltoAll2DHGetBuffSize,
  AlltoAll2DHRunTest
};

#pragma weak ncclTestEngine=alltoAll2DHEngine
